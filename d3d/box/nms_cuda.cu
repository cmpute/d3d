#include "hip/hip_runtime.h"
#include <d3d/common.h>
#include <d3d/box/nms.h>
#include <d3d/box/geometry.hpp>

using namespace std;
using namespace torch;

constexpr int FlagWidth = sizeof(int64_t) * 8;

// FIXME: Is there any reason to cut blocks like this? Why not directly calculate?
// FIXME: Should have quicker solution, directly compare each pair boxes and suppress the box with
//        lower score if they have overlap greater than threshold.
//        This should be only considered if it takes too much time with respect to whole process.

template <typename scalar_t>
__global__ void rbox_2d_nms_kernel(
    const _PackedAccessor(2) boxes,
    const _PackedAccessorT(int64_t, 1) order,
    const scalar_t threshold,
    _PackedAccessorT(int64_t, 2) mask
) {
    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;
    if (row_start > col_start) return; // calculate only blocks in upper triangle part

    const int row_size = min(boxes.size(0) - row_start * FlagWidth, FlagWidth);
    const int col_size = min(boxes.size(0) - col_start * FlagWidth, FlagWidth);

    __shared__ scalar_t block_boxes[FlagWidth][5]; // XXX: find a way to declare Box2 object here directly
    if (threadIdx.x < col_size)
    {
        #pragma unroll
        for (int i = 0; i < 5; ++i)
        {
            int boxi = order[FlagWidth * col_start + threadIdx.x];
            block_boxes[threadIdx.x][i] = boxes[boxi][i];
        }
    }
    __syncthreads();

    // calculate suppression in this cropped box
    if (threadIdx.x < row_size)
    {
        const int idx = FlagWidth * row_start + threadIdx.x;
        const int bcur_idx = order[idx];
        Box2 bcur(boxes[bcur_idx][0], boxes[bcur_idx][1], boxes[bcur_idx][2],
            boxes[bcur_idx][3], boxes[bcur_idx][4]);

        int64_t flag = 0;
        int start = (row_start == col_start) ? threadIdx.x + 1 : 0; // also calculate only upper part in diagonal blocks
        for (int i = start; i < col_size; i++)
        {
            Box2 bcomp(block_boxes[i][0], block_boxes[i][1], block_boxes[i][2],
                block_boxes[i][3], block_boxes[i][4]);
            if (bcur.iou(bcomp) > threshold)
                flag |= 1ULL << i;
        }
        mask[idx][col_start] = flag;
    }
}

__global__ void nms_collect(
    const _PackedAccessorT(int64_t, 1) order,
    const _PackedAccessorT(int64_t, 2) mask,
    _PackedAccessorT(int64_t, 1) remv,
    _PackedAccessorT(bool, 1) suppressed // need to be filled by false
) {
    const int nboxes = mask.size(0);
    const int nblocks = mask.size(1);

    for (int i = 0; i < nboxes; i++)
    {
        int block_idx = i / FlagWidth;
        int thread_idx = i % FlagWidth;

        if (remv[block_idx] & (1ULL << thread_idx))
            suppressed[order[i]] = true;
        else // suppress succeeding blocks
            for (int j = block_idx; j < nblocks; j++)
                remv[j] |= mask[i][j];
    }
}

void rbox_2d_nms_cuda(
  const Tensor boxes, const Tensor order,
  float threshold,
  Tensor suppressed
) {
    const int nboxes = boxes.sizes().at(0);
    const int nblocks = DivUp(nboxes, FlagWidth);
    auto long_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kLong);

    // This tensor store pairwise IOU result, rows are continuous while cols are divided by FlagWidth.
    // It has type int64, but it can act as uint64 in terms of bit operation.
    // Also note that the index in mask is corresponding to the position in `order` tensor.
    auto mask = torch::zeros({nboxes, nblocks}, long_options);

    dim3 blocks(nblocks, nblocks);
    dim3 threads(FlagWidth);

    AT_DISPATCH_FLOATING_TYPES(boxes.type(), "rbox_2d_nms_kernel", ([&] {
        rbox_2d_nms_kernel<<<blocks, threads>>>(
            boxes._packed_accessor(2),
            order._packed_accessor_typed(int64_t, 1),
            (scalar_t) threshold,
            mask._packed_accessor_typed(int64_t, 2));
    }));

    auto remv = torch::zeros({nblocks}, long_options); // suppression flags
    nms_collect<<<1, 1>>>(
        order._packed_accessor_typed(int64_t, 1),
        mask._packed_accessor_typed(int64_t, 2),
        remv._packed_accessor_typed(int64_t, 1),
        suppressed._packed_accessor_typed(bool, 1));
}
