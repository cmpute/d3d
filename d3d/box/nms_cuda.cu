#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

// Hard-coded maximum. Increase if needed.
#define MAX_COL_BLOCKS 1000

#define DIVUP(m,n) (((m)+(n)-1) / (n))
int64_t const threadsPerBlock = sizeof(unsigned long long) * 8;

// The functions below originates from Fast R-CNN
// See https://github.com/rbgirshick/py-faster-rcnn
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License
// Written by Shaoqing Ren

template <typename scalar_t>
__device__ inline scalar_t devIoU(scalar_t const * const a, scalar_t const * const b) {
  scalar_t left = max(a[0], b[0]), right = min(a[2], b[2]);
  scalar_t top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  scalar_t width = max(right - left, 0.f), height = max(bottom - top, 0.f);
  scalar_t interS = width * height;
  scalar_t Sa = (a[2] - a[0]) * (a[3] - a[1]);
  scalar_t Sb = (b[2] - b[0]) * (b[3] - b[1]);
  return interS / (Sa + Sb - interS);
}

template <typename scalar_t>
__global__ void nms_kernel(const int64_t n_boxes, const scalar_t nms_overlap_thresh,
                           const scalar_t *dev_boxes, const int64_t *idx, int64_t *dev_mask) {
  const int64_t row_start = blockIdx.y;
  const int64_t col_start = blockIdx.x;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ scalar_t block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const scalar_t *cur_box = dev_boxes + idx[cur_box_idx] * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 4) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


__global__ void nms_collect(const int64_t boxes_num, const int64_t col_blocks, int64_t top_k, const int64_t *idx, const int64_t *mask, int64_t *keep, int64_t *parent_object_index, int64_t *num_to_keep) {
  int64_t remv[MAX_COL_BLOCKS];
  int64_t num_to_keep_ = 0;

  for (int i = 0; i < col_blocks; i++) {
      remv[i] = 0;
  }

  for (int i = 0; i < boxes_num; ++i) {
      parent_object_index[i] = 0;
  }

  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;


    if (!(remv[nblock] & (1ULL << inblock))) {
      int64_t idxi = idx[i];
      keep[num_to_keep_] = idxi;
      const int64_t *p = &mask[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
      for (int j = i; j < boxes_num; j++) {
        int nblockj = j / threadsPerBlock;
        int inblockj = j % threadsPerBlock;
        if (p[nblockj] & (1ULL << inblockj))
            parent_object_index[idx[j]] = num_to_keep_+1;
      }
      parent_object_index[idx[i]] = num_to_keep_+1;

      num_to_keep_++;

      if (num_to_keep_==top_k)
          break;
    }
  }

  // Initialize the rest of the keep array to avoid uninitialized values.
  for (int i = num_to_keep_; i < boxes_num; ++i)
      keep[i] = 0;

  *num_to_keep = min(top_k,num_to_keep_);
}

#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")

std::vector<at::Tensor> nms_cuda_forward(
        at::Tensor boxes,
        at::Tensor idx,
        float nms_overlap_thresh,
        unsigned long top_k) {

  const auto boxes_num = boxes.size(0);

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  AT_ASSERTM (col_blocks < MAX_COL_BLOCKS, "The number of column blocks must be less than MAX_COL_BLOCKS. Increase the MAX_COL_BLOCKS constant if needed.");

  auto longOptions = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kLong);
  auto mask = at::empty({boxes_num * col_blocks}, longOptions);

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);

  CHECK_CONTIGUOUS(boxes);
  CHECK_CONTIGUOUS(idx);
  CHECK_CONTIGUOUS(mask);

  AT_DISPATCH_FLOATING_TYPES(boxes.type(), "nms_cuda_forward", ([&] {
    nms_kernel<<<blocks, threads>>>(boxes_num,
                                    (scalar_t)nms_overlap_thresh,
                                    boxes.data<scalar_t>(),
                                    idx.data<int64_t>(),
                                    mask.data<int64_t>());
  }));

  auto keep = at::empty({boxes_num}, longOptions);
  auto parent_object_index = at::empty({boxes_num}, longOptions);
  auto num_to_keep = at::empty({}, longOptions);

  nms_collect<<<1, 1>>>(boxes_num, col_blocks, top_k,
                        idx.data<int64_t>(),
                        mask.data<int64_t>(),
                        keep.data<int64_t>(),
                        parent_object_index.data<int64_t>(),
                        num_to_keep.data<int64_t>());


  return {keep,num_to_keep,parent_object_index};
}

