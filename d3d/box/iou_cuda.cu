#include "hip/hip_runtime.h"
#include <d3d/common.h>
#include <d3d/box/iou.h>
#include <d3d/box/geometry.hpp>

using namespace std;
using namespace torch;

template <typename scalar_t>
__global__ void rbox_2d_iou_kernel(
    const _PackedAccessor(2) boxes1,
    const _PackedAccessor(2) boxes2,
    _PackedAccessor(2) ious
) {
    const int nm = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = nm / boxes1.size(0);
    const int j = nm % boxes1.size(0);
    
    Box2 bi(boxes1[i][0], boxes1[i][1], boxes1[i][2],
        boxes1[i][3], boxes1[i][4]);
    Box2 bj(boxes2[j][0], boxes2[j][1], boxes2[j][2],
        boxes2[j][3], boxes2[j][4]);
    ious[i][j] = bi.iou(bj);
}

void rbox_2d_iou_cuda(
    const Tensor boxes1, const Tensor boxes2, Tensor ious
) {
    const int total_ops = boxes1.sizes().at(0) * boxes2.sizes().at(0);
    const int threads = 1024;
    const dim3 blocks(DivUp(total_ops, threads));
  
    AT_DISPATCH_FLOATING_TYPES(boxes1.type(), "rbox_2d_iou_cuda", ([&] {
        rbox_2d_iou_kernel<scalar_t><<<blocks, threads>>>(
        boxes1._packed_accessor(2),
        boxes2._packed_accessor(2),
        ious._packed_accessor(2));
    }));
}
